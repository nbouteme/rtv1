#include "hip/hip_runtime.h"
#include "ctypes.hu"

__host__
float deg2rad(float deg)
{
	const float tran = 180.0f / M_PI;

	return deg / tran;
}

__host__
t_primitive *new_primitive(t_primitive *alloc, t_iprimitive *base)
{
	t_cvec3 rot_axis = t_cvec3(base->trans.rot_axis).normalize();
	alloc->transform = translate(base->trans.pos);
	alloc->transform = alloc->transform * rotate(rot_axis, deg2rad(base->trans.rot_angle));
	//* t_cmat4(1);
	alloc->itransform = alloc->transform.inverse();
	alloc->norm = alloc->itransform.topleft().transpose();
	alloc->inorm = alloc->transform.topleft().transpose();
	alloc->diffuse = t_cvec3(base->mat.diffuse);
	alloc->ambiant = t_cvec3(base->mat.ambiant);
	alloc->spec = base->mat.spec_intensity;
	return (alloc);
}
