#include "hip/hip_runtime.h"
#include "ctypes.hu"

__host__
float deg2rad(float deg)
{
	const float tran = 180.0f / M_PI;

	return deg / tran;
}

__host__
t_primitive *new_primitive(t_primitive *alloc, t_iprimitive *base)
{
	t_cvec3 rot_axis = t_cvec3(base->trans.rot_axis).normalize();
	alloc->transform = translate(base->trans.pos);
	alloc->transform = alloc->transform * rotate(rot_axis, deg2rad(base->trans.rot_angle));
	alloc->itransform = alloc->transform.inverse();
	alloc->norm = alloc->itransform.topleft().transpose();
	alloc->inorm = alloc->transform.topleft().transpose();
	alloc->mat.ambiant = base->mat.ambiant;
	alloc->mat.diffuse = base->mat.diffuse;
	alloc->mat.spec_intensity = base->mat.spec_intensity;
	alloc->mat.reflectivity = base->mat.reflectivity;
	alloc->mat.transluscence = base->mat.transluscence;
	return (alloc);
}
