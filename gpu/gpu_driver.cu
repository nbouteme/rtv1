#include "hip/hip_runtime.h"
extern "C" {
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <cuda/hip/hip_runtime.h>
#include <xmlx.h>
}

#include <ctime>
#include "ctypes.hu"
#include "scene.hu"

__global__
void draw_scene(t_cuda_scene &scene);
__host__ __device__
void cuda_bake_camray(t_ccamera *c);

hipGraphicsResource *vbo_res;
surface<void, cudaSurfaceType2D> surRef;
int tex;

#include "device_code.cuda"

void move_camera(cuda_context *cc, int key)
{
	t_cvec3 &pos = cc->scene->cam.pos;
	t_cvec3 &forward = cc->scene->cam.dir;
	t_cvec3 left = cc->scene->cam.u;
	left.normalize();
	t_cvec3 &up = cc->scene->cam.up;
	if (key == XMLX_KEY_LEFT)
		pos = pos + left;
	else if (key == XMLX_KEY_RIGHT)
		pos = pos - left;
	else if (key == XMLX_KEY_UP)
		pos = pos + forward;
	else if (key == XMLX_KEY_DOWN)
		pos = pos - forward;
	else if (key == XMLX_KEY_KP_SUBTRACT)
		pos = pos + up;
	else if (key == XMLX_KEY_KP_ADD)
		pos = pos - up;
	//hipMemcpy(cc->gpu_scene, cc->scene, cc->scene->size, hipMemcpyHostToDevice);
}

void rotate_camera(cuda_context *cc, double x, double y)
{
	static int oldx = 0, oldy = 0;
	// si le bouton de la souris est enfoncé
	if(cc->mouse_state[0] == 1)
	{
		// calcule la difference d'angle par pixel pour un champs de vision de 45 deg
		float xangle = 3.141529f / 4 / 1280;
		float yangle = 3.141529f / 4 / 720;

		t_cvec3 left = cc->scene->cam.u;
		left.normalize();
		t_cvec3 &up = cc->scene->cam.up;
		t_cvec3 &dir = cc->scene->cam.dir;

		dir = rotate(dir, float(y - oldy) * -yangle, left);
		dir = rotate(dir, float(x - oldx) * xangle, up);
		up = rotate(up, float(y - oldy) * -yangle, left);
	}
	oldx = x, oldy = y;
}

extern "C"
int cuda_genimage(t_display *disp)
{
	//static int texinit = 0;
	cuda_context *cc = (cuda_context*)disp->renderer_driver->ctx;
	cuda_bake_camray(&cc->scene->cam);

	cc->scene->primitives = (t_primitive*)(((char*)cc->gpu_scene) + sizeof(*cc->scene));
	cc->scene->spots = (t_cspot*)(((char*)cc->scene->primitives) + sizeof(t_primitive) * cc->scene->n_primitives);

	hipMemcpy(cc->gpu_scene, cc->scene, cc->scene->size, hipMemcpyHostToDevice);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}

	hipArray *array;
    dim3 block(8, 48, 1);
    dim3 grid(1280 / 32 / block.x, 720 / block.y, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipGraphicsMapResources(1, &vbo_res, 0);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	hipGraphicsSubResourceGetMappedArray(&array, vbo_res, 0, 0);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	cudaBindSurfaceToArray(surRef, array);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	hipEventRecord(start);
	
	draw_scene<<<grid, block>>>(*cc->gpu_scene);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	hipEventRecord(stop);
    hipDeviceSynchronize();
	hipEventSynchronize(stop);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	hipGraphicsUnmapResources(1, &vbo_res);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("\r%f ms, %f fps.", milliseconds, 1.0f / (milliseconds / 1000.0f));
	fflush(stdout);
	return 0;
}

extern "C"
void cuda_fini(t_driver *driver)
{
	hipDeviceReset();
}

void set_mouse_state(cuda_context *cc, int button, int act, int mod)
{
	(void)mod;
	cc->mouse_state[button] = act;
}

extern "C"
void cuda_init(t_display *disp)
{
	disp->key_handler = (t_keyhandle_fun) move_camera;
	disp->mouse_press_handler = (t_mousehandle_fun)set_mouse_state;
	disp->mouse_handler = (t_mousehandle_fun)rotate_camera;
	cuda_context *cc = new cuda_context;
	cc->mouse_state[0] = 0;
	tex = (int)(long)disp->disp_param;
	disp->renderer_driver->ctx = (void*)cc;

	vbo_res = 0;
	cudaGLSetGLDevice(0);
	glBindTexture(GL_TEXTURE_2D, tex);
	hipGraphicsGLRegisterImage(&vbo_res, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
	cc->scene = generate_cscene((t_iscene*)disp->user_ptr);
	cc->gpu_scene = 0;
	auto e = hipMalloc(&cc->gpu_scene, 4096);//cc->scene->size);
}

t_primitive *new_primitive(t_primitive *alloc, t_iprimitive *base);

#define D2(a, b) a ## b
#define D(x) D2(x, _cuda_ray_intersect)

t_cuda_scene *generate_cscene(t_iscene *fn)
{
	t_ray_intersect_fun funs[4];	

	hipMemcpyFromSymbol(&funs[0], HIP_SYMBOL(I(plane)), sizeof(void*));
	hipMemcpyFromSymbol(&funs[1], HIP_SYMBOL(I(sphere)), sizeof(void*));
	hipMemcpyFromSymbol(&funs[2], HIP_SYMBOL(I(cone)), sizeof(void*));
	hipMemcpyFromSymbol(&funs[3], HIP_SYMBOL(I(cylinder)), sizeof(void*));
	int i;
	i = 0;

	t_cuda_scene *ret;
	ret = (t_cuda_scene*)malloc(sizeof(t_cuda_scene) +
								sizeof(t_primitive) * fn->n_primitives +
								sizeof(t_spot) * fn->n_spots);
	memcpy(ret->cam.dir.v, fn->cam_dir, sizeof(t_vec3));
	memcpy(ret->cam.pos.v, fn->cam_pos, sizeof(t_vec3));
	memcpy(ret->cam.up.v, fn->cam_up, sizeof(t_vec3));
	ret->primitives = (t_primitive*)(((char*)ret) + sizeof(*ret));
	ret->spots = (t_cspot*)(((char*)ret->primitives) + sizeof(t_primitive) * fn->n_primitives);
	ret->n_primitives = fn->n_primitives;
	ret->n_spots = fn->n_spots;
	ft_memcpy(&ret->spots[0], fn->spots, sizeof(t_spot) * fn->n_spots);
	while (i < fn->n_primitives)
	{
		new_primitive(&ret->primitives[i], &fn->primitives[i]);
		ret->primitives[i].type = fn->primitives[i].type;
		++i;
	}
	ret->size = sizeof(t_cuda_scene) +
		sizeof(t_primitive) * fn->n_primitives +
		sizeof(t_spot) * fn->n_spots;
	return (ret);
}
