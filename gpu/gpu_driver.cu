#include "hip/hip_runtime.h"
extern "C" {
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>
#include <cuda/hip/hip_runtime.h>
}

#include "ctypes.hu"
#include "scene.hu"

__global__
void draw_scene(t_cuda_scene &scene);
__host__ __device__
void cuda_bake_camray(t_ccamera *c);

hipGraphicsResource *vbo_res;
surface<void, cudaSurfaceType2D> surRef;
int tex;

#include "device_code.cuda"

extern "C"
void cuda_genimage(t_display *disp)
{
	//static int texinit = 0;
	cuda_context *cc = (cuda_context*)disp->renderer_driver->ctx;
	cuda_bake_camray(&cc->scene->cam);

	hipMemcpy(cc->gpu_scene, cc->scene, cc->scene->size, hipMemcpyHostToDevice);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}

	hipArray *array;
    dim3 block(8, 48, 1);
    dim3 grid(1280 / block.x, 720 / block.y, 1);
	hipGraphicsMapResources(1, &vbo_res, 0);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	hipGraphicsSubResourceGetMappedArray(&array, vbo_res, 0, 0);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	cudaBindSurfaceToArray(surRef, array);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	draw_scene<<<grid, block>>>(*cc->gpu_scene);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
    hipDeviceSynchronize();
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
	hipGraphicsUnmapResources(1, &vbo_res);
	{
		hipError_t err = hipGetLastError();
		if (err != hipSuccess) 
			printf("Error: %d: %s\n", __LINE__, hipGetErrorString(err));
	}
}

extern "C"
void cuda_fini(t_driver *driver)
{
	hipDeviceReset();
}

extern "C"
void cuda_init(t_display *disp)
{
	cuda_context *cc = new cuda_context;
	tex = (int)(long)disp->disp_param;
	disp->renderer_driver->ctx = (void*)cc;

	vbo_res = 0;
	cudaGLSetGLDevice(0);
	glBindTexture(GL_TEXTURE_2D, tex);
	hipGraphicsGLRegisterImage(&vbo_res, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
	// Par rapport a la scene sur cpu, la scene sur GPU est plate
	//cc->scene = generate_cscene((t_iscene*)disp->user_ptr);
	cc->scene = generate_cscene((t_iscene*)disp->user_ptr);
	printf("SIZE: %d\n", cc->scene->size);
	cc->gpu_scene = 0;
	auto e = hipMalloc(&cc->gpu_scene, 4096);//cc->scene->size);
	printf("Allocated scene, size: %p, %d\n", cc->gpu_scene, cc->scene->size);
}

t_primitive *new_primitive(t_primitive *alloc, t_iprimitive *base);

#define D2(a, b) a ## b
#define D(x) D2(x, _cuda_ray_intersect)

t_cuda_scene *generate_cscene(t_iscene *fn)
{
	t_ray_intersect_fun funs[4];	

	hipMemcpyFromSymbol(&funs[0], HIP_SYMBOL(I(plane)), sizeof(void*));
	hipMemcpyFromSymbol(&funs[1], HIP_SYMBOL(I(sphere)), sizeof(void*));
	hipMemcpyFromSymbol(&funs[2], HIP_SYMBOL(I(cone)), sizeof(void*));
	hipMemcpyFromSymbol(&funs[3], HIP_SYMBOL(I(cylinder)), sizeof(void*));
	int i;
	i = 0;

	t_cuda_scene *ret;
	ret = (t_cuda_scene*)malloc(sizeof(t_cuda_scene) +
								sizeof(t_primitive) * fn->n_primitives +
								sizeof(t_spot) * fn->n_spots);
	memcpy(ret->cam.dir.v, fn->cam_dir, sizeof(t_vec3));
	memcpy(ret->cam.pos.v, fn->cam_pos, sizeof(t_vec3));
	memcpy(ret->cam.up.v, fn->cam_up, sizeof(t_vec3));
	ret->primitives = (t_primitive*)(((char*)ret) + sizeof(*ret));
	ret->spots = (t_cspot*)(((char*)ret->primitives) + sizeof(t_primitive) * fn->n_primitives);
	ret->n_primitives = fn->n_primitives;
	ret->n_spots = fn->n_spots;
	ft_memcpy(&ret->spots[0], fn->spots, sizeof(t_spot) * fn->n_spots);
	while (i < fn->n_primitives)
	{
		new_primitive(&ret->primitives[i], &fn->primitives[i]);
		ret->primitives[i].intersect = funs[fn->primitives[i].type];
		printf("assigned %p\n", ret->primitives[i].intersect);
		++i;
	}
	ret->size = sizeof(t_cuda_scene) +
		sizeof(t_primitive) * fn->n_primitives +
		sizeof(t_spot) * fn->n_spots;
	return (ret);
}
